
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void check(long long to) {
    long long index = blockIdx.x * blockDim.x + threadIdx.x;
    long long stride = blockDim.x * gridDim.x;
    for (long long j = index; j < to; j += stride) {
        long long n = to-j;
        char *ans = "", *ans1 = "";
        bool can = 0;
        for (long long a = 2; a < n; ++a) {
            bool tf = 1;
            if (((a | n) == n && (a & n) == a)) {
                long long rest = n ^ a;
                if (rest == 1 && __popcll(a) > 1) {
                    rest += (a & -a);
                }
                if (rest < n && rest > 1 && a < n && ((rest | a) == n)) {
                    tf = 0, can = 1;
                    ans = "No";
                    break;
                }
            }
            if (!tf) {
                break;
            }
        }
        if (!can) {
            ans = "Yes";
        }
        if ((n & 1) && __popcll(n) <= 2) {
            ans1 = "Yes";
        } else if (__popcll(n) == 1) {
            ans1 = "Yes";
        } else {
            ans1 = "No";
        }
        if (n % 10000000 == 0) {
            printf("Completed for %llu\n", n);
        }
        if (ans != ans1) {
            printf("ERROR %llu\n", n);
            return;
        }
    }
}

int main() {
    long long to = 1e18;
    long long blockSize = 256;
    long long numBlocks = (to + blockSize - 1) / blockSize;
    check<<<1, blockSize>>>(to);
    hipDeviceSynchronize();
}
