
#include <hip/hip_runtime.h>
#include <fstream>

using namespace std;

__global__
void check(int n, long long *res){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int cur=index;cur<=n;cur+=stride){
        unsigned int s=cur;
        long long steps=1;
        do{
            s = (s * 1103515245 + 12345);
            steps++;
        }while(s!=cur);
        res[cur]=steps;
        if(cur%10000==0){
            printf("Completed %d\n", cur);
        }
    }
}

int main(){
    int n = 1e5;
    long long *res=new long long(n+1);
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    hipMallocManaged(&res, (n+1)*sizeof(long long));
    check<<<numBlocks, blockSize>>>(n, res);
    hipDeviceSynchronize();
    ofstream out("output.txt");
    for(int i=0;i<=n;i++){
        out<<i<<" cycle: "<<res[i]<<"\n";
    }
    hipFree(res);
    return 0;
}