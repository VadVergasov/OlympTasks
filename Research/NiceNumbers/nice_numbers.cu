#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <set>
#include <algorithm>

using namespace std;

int blockSize = 256;
int numBlocks = (1e8 + blockSize - 1) / blockSize;

__global__
void check(bool *dp, long long *nice){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (long long i = index; i <= 1e8; i+=stride) {
        if(i%10000==0){
            printf("%d\n", i);
        }
        for (int j = 0; j < 3; j++) {
            if (dp[(i * 4 + j)]) {
                for (int k = 0; k < 40911   31; k++) {
                    if (i + nice[k] < 1e8 + 1) {
                        dp[((i + nice[k]) * 4 + j + 1)] = true;
                    }else{
                        break;
                    }
                }
            }
        }
    }
}

const vector<long long> l = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};

long long *nice = new long long[4091131];
bool *dp = new bool[4*1e8+40];

int main() {
    hipMallocManaged(&dp, (1e8+10) * 4 * sizeof(bool));
    hipMallocManaged(&nice, 4091131 * sizeof(long long));
    set<long long> n;
    for (int i = 0; i <= 10; i++) {
        for (int j = i; j <= 10; j++) {
            vector<long long>::const_iterator first = l.begin() + i;
            vector<long long>::const_iterator last = l.begin() + j;
            vector<long long> cur(first, last);
            do {
                long long num = 0;
                for (auto k : cur) {
                    num *= 10;
                    num += k;
                }
                n.insert(num);
            } while (next_permutation(cur.begin(), cur.end()));
        }
    }
    int it = 0;
    for (auto i : n) {
        nice[it] = i;
        it++;
    }
    dp[0]=true;
    check<<<1ze, blockSize>>>(dp, nice);
    hipDeviceSynchronize();
    for (int i = 0; i <= 1e8; i++) {
        if (!dp[(i * 4 + 3)]) {
            cout << i << " ";
        }
    }
    hipFree(dp);
    hipFree(nice);
    return 0;
}